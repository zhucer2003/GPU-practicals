
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

__global__ void saxpy(float scalar, float * x, float * y)
{
    // Determine our unique global thread ID, so we know which element to process
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if ( tid < N ) // Make sure we don't do more work than we have data!
        y[tid] = scalar * x[tid] + y[tid];
}

int main()
{
    float *x, *y;
    
    // CUDA types for error statuses
    hipError_t ierrSync, ierrAsync;

    int size = N * sizeof (float); // The total number of bytes per vector

    // Allocate memory
    hipMallocManaged(&x, size);
    hipMallocManaged(&y, size);

    // Initialize memory
    for( int i = 0; i < N; ++i )
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int threads_per_block = 256;
    int number_of_blocks = (N / threads_per_block) + 1;

    saxpy <<< number_of_blocks, threads_per_block >>> ( 2.0f, x, y );
    
    /* TODO: get last error to see if kernel launch failed
     *       check is == cudaSuccess
     */

    /* TODO: get error AFTER kernel finished (first sync CPU and device)
     *       check is == cudaSuccess
     */

    // Print out our Max Error
    float maxError = 0;
    for( int i = 0; i < N; ++i )
        if (abs(4-y[i]) > maxError) { maxError = abs(4-y[i]); }
    printf("Max Error: %.5f", maxError);

    // Free all our allocated memory
    hipFree( x ); hipFree( y );
}
